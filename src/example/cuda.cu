#include "hip/hip_runtime.h"
#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>


namespace nb = nanobind;
using namespace nb::literals;


template <typename Scalar>
__global__ void double_arr_kernel(Scalar* out, const Scalar* in, size_t size) {
    size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        out[i] = 2 * in[i];
    }
}


template <typename Scalar>
void double_arr(nb::ndarray<Scalar, nb::ndim<1>, nb::device::cuda> outarr,
            nb::ndarray<const Scalar, nb::ndim<1>, nb::device::cuda> inarr) {

    size_t size = inarr.size();
    size_t block_size = 256;
    size_t num_blocks = (size + block_size - 1) / block_size;

    double_arr_kernel<<<num_blocks, block_size>>>(outarr.data(), inarr.data(), size);
}


NB_MODULE(cuda, m) {
    m.def("double_arr",
        &double_arr<float>,
        "outarr"_a.noconvert(),
        "inarr"_a.noconvert()
        );
}
